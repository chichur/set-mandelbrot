#include "hip/hip_runtime.h"
#include <iostream>

void checkDevice();

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}


int main() {

    checkDevice();

    int N = 9;
    size_t size = sizeof(float) * N;

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size) ;
    float* h_C = (float*)malloc(size);

    float A[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    float B[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};

    memcpy(h_A, A, size);
    memcpy(h_B, B, size);

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    VecAdd<<<1, N>>>(d_A, d_B, d_C);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    float f = 1.0f;
    for (int i = 0; i < N; i++) {
        printf("%.0f %.0f %.0f\n", h_B[i], h_A[i], h_C[i]);
//        std::cout << h_A[i] << std::endl;
    }


    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

void checkDevice() {
    printf("-------------- Driver information --------------\n");

    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    int driver_version = 0, runtime_version = 0;

    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("Driver Version: %d\n"
           "Runtime Version: %d\n",
           driver_version, runtime_version);
    printf("------------------------------------------------\n");
}
